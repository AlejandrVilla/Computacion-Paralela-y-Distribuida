#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#define BLUR_SIZE 3
#define CHANNELS 3
#define uc unsigned char

using namespace cv;

__global__
void colorToBlurConversion(uc* out, uc* in, int w, int h) {
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if (Col < w && Row < h) {
		int pixValsr = 0;
		int pixValsg = 0;
		int pixValsb = 0;
		int pixels=0;
		int Offset = (Row * w + Col) * CHANNELS;

		for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
			for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
				int curRow = Row + blurRow;
				int curCol = Col + blurCol;
				if (curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
					int curOffset = (curRow * w + curCol) * CHANNELS;
					pixValsr += in[curOffset];
					pixValsg += in[curOffset + 1];
					pixValsb += in[curOffset + 2];
					pixels++;
				}
			}
		}
		out[Offset] = (uc)(pixValsr / pixels);
		out[Offset + 1] = (uc)(pixValsg / pixels);
		out[Offset + 2] = (uc)(pixValsb / pixels);
	}
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << "Uso: " << argv[0] << " <nombre_de_archivo_imagen>" << std::endl;
        return -1;
    }

    int width, height;
    Mat image1 = imread(argv[1], IMREAD_COLOR);

    if (image1.empty()) {
        std::cerr << "Error al cargar la imagen." << std::endl;
        return -1;
    }

    namedWindow("Imagen Original", WINDOW_AUTOSIZE);
    imshow("Imagen Original", image1);
    waitKey(0);

    Size imageSize = image1.size();
    width = imageSize.width;
    height = imageSize.height;

    std::cout<<width<<" "<<height<<"\n";

    uc* ptrImageData = NULL;
    uc* ptrImageDataOut = NULL;

    hipMalloc(&ptrImageDataOut, width * height * CHANNELS);
    hipMalloc(&ptrImageData, width * height * CHANNELS);
    hipMemcpy(ptrImageData, image1.data, width * height * CHANNELS, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(width / 16.0), ceil(height / 16.0), 1);
    dim3 dimBlock(16, 16, 1);

    colorToBlurConversion<<<dimGrid, dimBlock>>>(ptrImageDataOut, ptrImageData, width, height);
    hipDeviceSynchronize(); // Esperar a que todos los bloques terminen

    Mat image2(height, width, CV_8UC3);
    hipMemcpy(image2.data, ptrImageDataOut, width * height * CHANNELS, hipMemcpyDeviceToHost);

    std::string nuevoNombre = argv[1];
    nuevoNombre = nuevoNombre.substr(0, nuevoNombre.find_last_of('.')) + "_toBlur.jpeg";
    imwrite(nuevoNombre, image2);

    hipFree(ptrImageData);
    hipFree(ptrImageDataOut);

    namedWindow("Imagen Procesada", WINDOW_AUTOSIZE);
    imshow("Imagen Procesada", image2);
    waitKey(0);

    return 0;
}
